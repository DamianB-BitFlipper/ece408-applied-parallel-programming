#include "hip/hip_runtime.h"
// MP 5 Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ... + lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

__global__ void scanAddExclusive(float* input, float* output, int len) {
    //@@ Modify the body of this function to complete the functionality of
    //@@ the scan on the device
    //@@ You may need multiple kernel calls; write your kernels before this
    //@@ function and call them from here
    __shared__ float runningSum_sm[2 * BLOCK_SIZE];

    uint32_t tid{ threadIdx.x };

    // Whether to read in, write out from the 0th or 1st `BLOCK_SIZE` of `runningSum_sm`
    int32_t pout{ BLOCK_SIZE }, pin{ 0 };

    runningSum_sm[pin + tid] = (tid < len) ? input[tid] : 0;
    runningSum_sm[pout + tid] = 0;
    __syncthreads();

    for (int32_t stride{ 1 }; stride < len; stride *= 2) {
        // Add the value from `stride` before
        if (tid >= stride) {
            runningSum_sm[pout + tid] = runningSum_sm[pin + tid] + runningSum_sm[pin + tid - stride];
        } else {
            // Copy from `pin` to `pout`
            runningSum_sm[pout + tid] = runningSum_sm[pin + tid];
        }

        // Swap the `pout`  and `pin` locations for the next iteration
        pout = BLOCK_SIZE - pout;
        pin = BLOCK_SIZE - pin;

        __syncthreads();
    }

    // Coppy from the `runningSum_sm` to the `output`
    if (tid < len) {
        output[tid] = runningSum_sm[pin + tid];
    }

    return;
}

int main(int argc, char** argv) {
    wbArg_t args;
    float* hostInput; // The input 1D list
    float* hostOutput; // The output list
    float* deviceInput;
    float* deviceOutput;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float*)wbImport(wbArg_getInputFile(args, 0), &numElements);
    hostOutput = (float*)malloc(numElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc((void**)&deviceInput, numElements * sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceOutput, numElements * sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid{ 1, 1, 1 };
    dim3 DimBlock{ BLOCK_SIZE, 1, 1 };

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Modify this to complete the functionality of the scan
    //@@ on the deivce
    scanAddExclusive<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numElements);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(
        hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");

    hipFree(deviceInput);
    hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    free(hostOutput);

    return 0;
}

