#include "hip/hip_runtime.h"
// MP 4 Reduction
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include <wb.h>
#include <cstdint>

#define BLOCK_SIZE 512 //@@ You can change this
#define WARP_SIZE 32
#define WINDOW_SIZE_PER_BLOCK 4

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

template <uint32_t blockSize>
__device__ void warpSum(volatile float* partial_sum, uint32_t tid) {
    // Unroll the last warp of summing computation
    if (blockSize >= 32) {
        partial_sum[tid] += partial_sum[tid + 32];
    }
    if (blockSize >= 16) {
        partial_sum[tid] += partial_sum[tid + 16];
    }
    if (blockSize >= 8) {
        partial_sum[tid] += partial_sum[tid + 8];
    }
    if (blockSize >= 4) {
        partial_sum[tid] += partial_sum[tid + 4];
    }
    if (blockSize >= 2) {
        partial_sum[tid] += partial_sum[tid + 2];
    }
    if (blockSize >= 1) {
        partial_sum[tid] += partial_sum[tid + 1];
    }
    return;
}

template <uint32_t blockSize>
__global__ void sum(float* input, float* output, int len) {
    //@@ Load a segment of the input vector into shared memory
    //@@ Traverse the reduction tree
    //@@ Write the computed sum of the block to the output vector at the 
    //@@ correct index
    __shared__ float partial_sum[2 * BLOCK_SIZE];

    uint32_t tid = threadIdx.x;
    uint32_t start = WINDOW_SIZE_PER_BLOCK * blockIdx.x * blockDim.x;

    // Each thread loads 4 values from global memory. It performs one sum outside
    // of the for-loop and stores the resulting 2 values in shared memory
    uint32_t input_loc0 = start + tid;
    bool load_loc0 = input_loc0 < len;
    uint32_t input_loc1 = input_loc0 + blockDim.x;
    bool load_loc1 = input_loc1 < len;
    uint32_t input_loc2 = input_loc1 + blockDim.x;
    bool load_loc2 = input_loc2 < len;
    uint32_t input_loc3 = input_loc2 + blockDim.x;
    bool load_loc3 = input_loc3 < len;

    // Some boundary checking
    if (load_loc0 && load_loc1) {
        partial_sum[2 * tid] = input[input_loc0] + input[input_loc1];
    } else if (load_loc0 && !load_loc1) {
        partial_sum[2 * tid] = input[input_loc0];
    } else {
        partial_sum[2 * tid] = 0;
    }

    // Some boundary checking
    if (load_loc2 && load_loc3) {
        partial_sum[2 * tid + 1] = input[input_loc2] + input[input_loc3];
    } else if (load_loc2 && !load_loc3) {
        partial_sum[2 * tid + 1] = input[input_loc2];
    } else {
        partial_sum[2 * tid + 1] = 0;
    }

    // Wait for all threads to load their respective data
    __syncthreads();

    if (blockSize >= 1024) {
        // tid < 1024 is always true due to hardware limitations, no need to check
        partial_sum[tid] += partial_sum[tid + 1024];
        __syncthreads();
    }
    if (blockSize >= 512) {
        if (tid < 512) {
            partial_sum[tid] += partial_sum[tid + 512];
        }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 256) {
            partial_sum[tid] += partial_sum[tid + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 128) {
            partial_sum[tid] += partial_sum[tid + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 64) {
        if (tid < 64) {
            partial_sum[tid] += partial_sum[tid + 64];
        }
        __syncthreads();
    }

    // Perform the last bit of summation on the final warp with no need to
    // synchronize since it all executes in one warp anyway
    if (tid < WARP_SIZE) {
        warpSum<blockSize>(partial_sum, tid);
    }

    // The final partial sum is located at `partial_sum[0]`
    if (tid == 0) {
        output[blockIdx.x] = partial_sum[0];
    }
}

int main(int argc, char** argv) {
    wbArg_t args;
    float* hostInput; // The input 1D list
    float hostOutput; // The output value
    float* deviceInput;
    float* deviceOutput;
    int numInputElements; // number of elements in the input list
    int numOutputElements; // number of elements in the output list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float*)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

    numOutputElements = numInputElements / (WINDOW_SIZE_PER_BLOCK * BLOCK_SIZE);
    if (numInputElements % (WINDOW_SIZE_PER_BLOCK * BLOCK_SIZE)) {
        numOutputElements++;
    }

    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numInputElements);
    wbLog(TRACE, "The number of output elements in the input is ", numOutputElements);

    wbTime_start(GPU, "Allocating GPU memory.");

    //@@ Allocate GPU memory here
    hipMalloc(&deviceInput, numInputElements * sizeof(float));
    hipMalloc(&deviceOutput, numOutputElements * sizeof(float));

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");

    //@@ Copy memory to the GPU here
    hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);

    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(numOutputElements, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    wbTime_start(Compute, "Performing sum aggregation computation");

    //@@ Launch the GPU Kernel here
    sum<BLOCK_SIZE><<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);
    sum<BLOCK_SIZE><<<1, DimBlock>>>(deviceOutput, deviceOutput, numOutputElements);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");

    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(&hostOutput, deviceOutput, sizeof(float), hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");


    wbTime_stop(Compute, "Performing sum aggregation computation");

    wbTime_start(GPU, "Freeing GPU Memory");

    //@@ Free the GPU memory here
    hipFree(deviceInput);
    hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, &hostOutput, 1);

    free(hostInput);

    return 0;
}

