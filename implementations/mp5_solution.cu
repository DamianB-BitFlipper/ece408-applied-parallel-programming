#include "hip/hip_runtime.h"
// MP 5 Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ... + lst[n-1]}

#include <wb.h>
#include <cstdint>
#include <iostream>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

__device__ void scanAddUpSweep(
    volatile float* runningSum_sm,
    int32_t len,
    int32_t& pout,
    int32_t& pin
) {
    uint32_t tid{ threadIdx.x };

    for (int32_t stride{ 1 }; stride < len; stride *= 2) {
        int32_t threadMod{ stride * 2 };
        // If the current thread is on a strided position, add the stride amounts
        if ((tid + 1) % threadMod == 0) {
            runningSum_sm[pout + tid] =
                runningSum_sm[pin + tid - stride] + runningSum_sm[pin + tid];
        } else {
            // Simply copy the value over
            runningSum_sm[pout + tid] = runningSum_sm[pin + tid];
        }

        // Swap the `pout`  and `pin` locations for the next iteration
        pout = BLOCK_SIZE - pout;
        pin = BLOCK_SIZE - pin;

        __syncthreads();
    }
}

__device__ void scanAddDownSweep(
    volatile float* runningSum_sm,
    int32_t len,
    int32_t& pout,
    int32_t& pin
) {
    uint32_t tid{ threadIdx.x };

    for (int32_t stride{ len / 2 }, i{ 1 }; stride > 1; stride /= 2, i++) {
        int32_t lookBack{ stride / 2 };
        // If the current thread is on a strided position, add the stride amounts
        if (tid > lookBack && (tid + 1 - lookBack) % stride == 0) {
            runningSum_sm[pout + tid] =
                runningSum_sm[pin + tid] + runningSum_sm[pin + tid - lookBack];
        } else {
            // Simply copy the value over
            runningSum_sm[pout + tid] = runningSum_sm[pin + tid];
        }

        // Swap the `pout`  and `pin` locations for the next iteration
        pout = BLOCK_SIZE - pout;
        pin = BLOCK_SIZE - pin;

        __syncthreads();
    }
}

__global__ void scanAddExclusive(float* input, float* output, int32_t len) {
    //@@ Modify the body of this function to complete the functionality of
    //@@ the scan on the device
    //@@ You may need multiple kernel calls; write your kernels before this
    //@@ function and call them from here
    extern __shared__ float runningSum_sm[];

    uint32_t tid{ threadIdx.x };

    // Whether to read in, write out from the 0th or 1st `BLOCK_SIZE` of `runningSum_sm`
    int32_t pout{ BLOCK_SIZE }, pin{ 0 };

    // Load the values from global memory into shared memory
    runningSum_sm[pin + tid] = (tid < len) ? input[tid] : 0;
    runningSum_sm[pout + tid] = 0;
    __syncthreads();

    // Run the up-sweep routine
    scanAddUpSweep(runningSum_sm, len, pout, pin);

    // Run the down-sweep routine
    scanAddDownSweep(runningSum_sm, len, pout, pin);

    // Copy from the `runningSum_sm` to the `output`
    if (tid < len) {
        output[tid] = runningSum_sm[pin + tid];
    }

    return;
}

int main(int argc, char** argv) {
    wbArg_t args;
    float* hostInput; // The input 1D list
    float* hostOutput; // The output list
    float* deviceInput;
    float* deviceOutput;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float*)wbImport(wbArg_getInputFile(args, 0), &numElements);
    hostOutput = (float*)malloc(numElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc((void**)&deviceInput, numElements * sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceOutput, numElements * sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid{ 1, 1, 1 };
    dim3 DimBlock{ BLOCK_SIZE, 1, 1 };

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Modify this to complete the functionality of the scan
    //@@ on the deivce
    scanAddExclusive<<<DimGrid, DimBlock, 2 * BLOCK_SIZE * sizeof(float)>>>(deviceInput, deviceOutput, numElements);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(
        hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");

    hipFree(deviceInput);
    hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    free(hostOutput);

    return 0;
}

