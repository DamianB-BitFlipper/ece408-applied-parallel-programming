#include "hip/hip_runtime.h"
// MP 5 Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ... + lst[n-1]}

#include <wb.h>
#include <cstdint>
#include <iostream>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

__device__ void scanAddUpSweep(volatile float* runningSum_sm, int32_t len) {
    /* Perform the up-sweep in-place.
     **/
    uint32_t tid{ threadIdx.x };

    for (int32_t validThreads{ len / 2 }, stride{ 1 };
         validThreads > 0;
         validThreads /= 2, stride *= 2) {

        if (tid < validThreads) {
            // There were `tid` threads consuming `2 * stride` space before the current thread
            int32_t offset{ static_cast<int32_t>(2 * stride * tid) };
            int32_t leftVal{ offset + stride - 1 };
            int32_t rightVal{ leftVal + stride };

            runningSum_sm[rightVal] += runningSum_sm[leftVal];
        }

        __syncthreads();
    }
}

__device__ void scanAddDownSweep(volatile float* runningSum_sm, int32_t len) {
    uint32_t tid{ threadIdx.x };

    for (int32_t validThreads{ 2 }, stride{ len / 2 };
         validThreads <= len / 2;
         validThreads *= 2, stride /= 2) {

        int32_t leftVal{ static_cast<int32_t>(stride * (tid + 1) - 1) };
        int32_t rightVal{ leftVal + stride / 2 };

        // The last `tid` is always invalid because `rightVal` is out of bounds
        if (tid < validThreads - 1) {
            runningSum_sm[rightVal] += runningSum_sm[leftVal];
        }

        __syncthreads();
    }
}

__global__ void scanAddExclusive(float* input, float* output, int32_t len) {
    //@@ Modify the body of this function to complete the functionality of
    //@@ the scan on the device
    //@@ You may need multiple kernel calls; write your kernels before this
    //@@ function and call them from here
    extern __shared__ float runningSum_sm[];

    uint32_t tid{ threadIdx.x };

    // Load the values from global memory into shared memory
    runningSum_sm[tid] = (tid < len) ? input[tid] : 0;
    __syncthreads();

    // Run the up-sweep routine
    scanAddUpSweep(runningSum_sm, len);

    // Run the down-sweep routine
    scanAddDownSweep(runningSum_sm, len);

    // Copy from the `runningSum_sm` to the `output`
    if (tid < len) {
        output[tid] = runningSum_sm[tid];
    }

    return;
}

int main(int argc, char** argv) {
    wbArg_t args;
    float* hostInput; // The input 1D list
    float* hostOutput; // The output list
    float* deviceInput;
    float* deviceOutput;
    int numElements; // number of elements in the list

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (float*)wbImport(wbArg_getInputFile(args, 0), &numElements);
    hostOutput = (float*)malloc(numElements * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The number of input elements in the input is ", numElements);

    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck(hipMalloc((void**)&deviceInput, numElements * sizeof(float)));
    wbCheck(hipMalloc((void**)&deviceOutput, numElements * sizeof(float)));
    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Clearing output memory.");
    wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
    wbTime_stop(GPU, "Clearing output memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice));
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid{ 1, 1, 1 };
    dim3 DimBlock{ BLOCK_SIZE, 1, 1 };

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Modify this to complete the functionality of the scan
    //@@ on the deivce
    scanAddExclusive<<<DimGrid, DimBlock, BLOCK_SIZE * sizeof(float)>>>(deviceInput, deviceOutput, numElements);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    wbCheck(
        hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost));
    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");

    hipFree(deviceInput);
    hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, numElements);

    free(hostInput);
    free(hostOutput);

    return 0;
}

